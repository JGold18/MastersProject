

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void addVector(int N, float *pt_a, float *pt_b, float *pt_c ){

  int i;
  for(i = 0; i < N; ++i){
    pt_c[i] = pt_a[i] - pt_b[i];
  }
  
}

int main () {

  /* specify number of entries */
  int N = 10;

  /* allocate enough memory to store N floats */
  /* malloc is a standard library call */
  /* malloc accepts one argument, specifying the number of bytes of memory to be  allocated */
  /* malloc does not initialize values in the array */
  float* pt_a = (float*) malloc(sizeof(float)*N);
  float* pt_b = (float*) malloc(sizeof(float)*N);
  float* pt_c = (float*) malloc(sizeof(float)*N);

  int i;

  for (i = 0; i < N; i=i+1){
    pt_a[i] = 7.7;
    pt_b[i] = 5.2;
  }

  addVector(N, pt_a, pt_b, pt_c);
  
  for  (i = 0; i < N; i++){  
    printf("c[i]=%f\n", pt_c[i]);	   
  }

  return 0;
						
}
