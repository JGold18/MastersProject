
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void addVector(int N, float* c_a, float* c_b, float* c_c ){

   int point_id = threadIdx.x +blockIdx.x*blockDim.x;
   if (point_id<N)
   {
        c_c[point_id] = c_a[point_id] - c_b[point_id];
   }
}

int main () {
 
  /* specify number of entries */
  int N = 10000;

  /* allocate enough memory to store N floats */
  /* malloc is a standard library call */
  /* malloc accepts one argument, specifying the number of bytes of memory to be  allocated */
  /* malloc does not initialize values in the array */
  float* pt_a = (float*) malloc(sizeof(float)*N);
  float* pt_b = (float*) malloc(sizeof(float)*N);
  float* pt_c = (float*) malloc(sizeof(float)*N);
  
  int i;

  for (i = 0; i < N; i=i+1){
    pt_a[i] = 7.7;
    pt_b[i] = 5.2;
  }

 //cuda mem allocation to gpu

    float* c_a;
    float* c_b;
    float* c_c;
 
    hipMalloc(&c_a, N*sizeof(float));
    hipMalloc(&c_b, N*sizeof(float));
    hipMalloc(&c_c, N*sizeof(float));
 
  //copying data from CPU(host) to GPU(Device)
 
    hipMemcpy(c_a, pt_a, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(c_b, pt_b, N*sizeof(float), hipMemcpyHostToDevice);


    int T=32;
    int B=(N + T-1)/T;
    dim3 NthreadsPerBlock(T);
    dim3 NBlocks(B);
    
    addVector<<<NBlocks,NthreadsPerBlock>>>(N, c_a, c_b, c_c);

//copying data from device to host
    hipMemcpy(pt_c, c_c, N*sizeof(float), hipMemcpyDeviceToHost);

  for(i = N - 100; i < N; i++){  
    printf("c[%d]=%f\n", i, pt_c[i]);	   
  }

  return 0;
						
}
